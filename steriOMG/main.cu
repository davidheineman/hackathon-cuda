#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

__global__ void map_tile(uchar4 *d_frame, int w, int h, int maxShift)
{
    if (blockIdx.y > h || blockIdx.x > w)
        return; // tile bounds
    uchar4 *outPx = d_frame + blockIdx.y * w * 2 + blockIdx.x + w;
    int offset = (*outPx).x * maxShift / 256;
    uchar4 *minInPx = d_frame + (blockIdx.y * w * 2);
    uchar4 *maxInPx = d_frame + (blockIdx.y * w * 2 + w - 1);
    uchar4 *inPx = outPx - w - offset;
    if (inPx < minInPx)
        inPx = minInPx;
    if (inPx > maxInPx)
        inPx = maxInPx;
    *outPx = *inPx;
}

int main()
{
    // Open the video file
    cv::VideoCapture cap("./fixed.mp4");
    if (!cap.isOpened())
    {
        std::cerr << "Error: Could not open the video file." << std::endl;
        return -1;
    }

    int width = cap.get(cv::CAP_PROP_FRAME_WIDTH);
    int height = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
    int totalFrames = cap.get(cv::CAP_PROP_FRAME_COUNT);
    int fps = cap.get(cv::CAP_PROP_FPS);

    // Create a VideoWriter object to save the processed video
    cv::VideoWriter writer("processed_video.mp4", cv::VideoWriter::fourcc('m', 'p', '4', 'v'), fps, cv::Size(width, height));

    // Allocate GPU memory for multiple frames
    const int numFrames = 10; // Process 10 frames per kernel launch
    uchar4 *d_frames;
    size_t frameSize = width * height * sizeof(uchar4);
    hipMalloc(&d_frames, frameSize * numFrames);

    // Allocate host memory for multiple frames
    std::vector<cv::Mat> frames(numFrames);
    std::vector<cv::Mat> framesRGBA(numFrames);

    // Process frames in batches
    for (int frameStart = 0; frameStart < totalFrames; frameStart += numFrames)
    {
        int framesInBatch = std::min(numFrames, totalFrames - frameStart);

        // Read frames
        for (int i = 0; i < framesInBatch; i++)
        {
            cap >> frames[i];
            if (frames[i].empty())
                break;
            cv::cvtColor(frames[i], framesRGBA[i], cv::COLOR_BGR2RGBA);
        }

        // Copy frames to GPU
        for (int i = 0; i < framesInBatch; i++)
        {
            hipMemcpy(d_frames + i * width * height, framesRGBA[i].data, frameSize, hipMemcpyHostToDevice);
        }

        // Call map_tile kernel for each frame
        dim3 blockSize(1, 1);
        dim3 gridSize(width / 2, height);
        int maxShift = 32; // Adjust this value as needed
        for (int i = 0; i < framesInBatch; i++)
        {
            map_tile<<<gridSize, blockSize>>>(d_frames + i * width * height, width / 2, height, maxShift);
        }

        // Copy results back to CPU
        for (int i = 0; i < framesInBatch; i++)
        {
            hipMemcpy(framesRGBA[i].data, d_frames + i * width * height, frameSize, hipMemcpyDeviceToHost);
        }

        // Convert back to BGR and write to output video
        for (int i = 0; i < framesInBatch; i++)
        {
            cv::Mat outputFrame;
            cv::cvtColor(framesRGBA[i], outputFrame, cv::COLOR_RGBA2BGR);
            writer.write(outputFrame);
        }
    }

    // Clean up
    hipFree(d_frames);
    cap.release();
    writer.release();

    return 0;
}